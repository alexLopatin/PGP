#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

void ReadArray(double* arr, int size);
void WriteArray(double* arr, int size);

void CudaOperation(double* first, double* second, double* result, int vectorLength);
__global__ void OperateParallel(double* firstDevice, double* secondDevice, double* resultDevice, int vectorLength);

__device__ double Min(double a, double b);

int main()
{
	int vectorLength;
	std::cin >> vectorLength;

	double* first;
	double* second;
	double* result;

	first = (double*)malloc(sizeof(double) * vectorLength);
	second = (double*)malloc(sizeof(double) * vectorLength);
	result = (double*)malloc(sizeof(double) * vectorLength);

	ReadArray(first, vectorLength);
	ReadArray(second, vectorLength);

	CudaOperation(first, second, result, vectorLength);

	WriteArray(result, vectorLength);
}

void ReadArray(double* arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		std::cin >> arr[i];
	}
}

void WriteArray(double* arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		std::cout << arr[i];

		if (i < size - 1)
		{
			std::cout << ' ';
		}
	}
}

void CudaOperation(double* first, double* second, double* result, int vectorLength)
{
	double* firstDevice = 0;
	double* secondDevice = 0;
	double* resultDevice = 0;

	hipSetDevice(0);

	hipMalloc((void**)&firstDevice, vectorLength * sizeof(double));
	hipMalloc((void**)&secondDevice, vectorLength * sizeof(double));
	hipMalloc((void**)&resultDevice, vectorLength * sizeof(double));

	hipMemcpy(firstDevice, first, vectorLength * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(secondDevice, second, vectorLength * sizeof(double), hipMemcpyHostToDevice);

	OperateParallel << <1, 256 >> > (firstDevice, secondDevice, resultDevice, vectorLength);
	hipDeviceSynchronize();

	hipMemcpy(result, resultDevice, vectorLength * sizeof(double), hipMemcpyDeviceToHost);
}

__device__ int Min(int a, int b)
{
	return (a > b)
		? b
		: a;
}

__device__ int Max(int a, int b)
{
	return (a < b)
		? b
		: a;
}

__global__ void OperateParallel(double* firstDevice, double* secondDevice, double* resultDevice, int vectorLength)
{
	int length = Max(vectorLength / blockDim.x, 1);
	int left = Min(threadIdx.x * length, vectorLength);
	int right = (threadIdx.x == blockDim.x - 1)
		? vectorLength - 1
		: Min(left + length - 1, vectorLength - 1);

	for (int i = left; i <= right; i++)
	{
		resultDevice[i] = Min(firstDevice[i], secondDevice[i]);
	}
}

__device__ double Min(double a, double b)
{
	return (a > b)
		? b
		: a;
}