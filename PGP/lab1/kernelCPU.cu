#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

void ReadArray(double* arr, int size);
void WriteArray(double* arr, int size);

void Operate(double* firstDevice, double* secondDevice, double* resultDevice, int vectorLength);

double Min(double a, double b);

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int vectorLength;
	std::cin >> vectorLength;

	double* first;
	double* second;
	double* result;

	first = (double*)malloc(sizeof(double) * vectorLength);
	second = (double*)malloc(sizeof(double) * vectorLength);
	result = (double*)malloc(sizeof(double) * vectorLength);

	ReadArray(first, vectorLength);
	ReadArray(second, vectorLength);

	hipEventRecord(start);

	Operate(first, second, result, vectorLength);

	hipEventRecord(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << milliseconds << std::endl;

	WriteArray(result, vectorLength);
}

void ReadArray(double* arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		std::cin >> arr[i];
	}
}

void WriteArray(double* arr, int size)
{
	for (int i = 0; i < size; i++)
	{
		std::cout << arr[i];

		if (i < size - 1)
		{
			std::cout << ' ';
		}
	}
}

int Min(int a, int b)
{
	return (a > b)
		? b
		: a;
}

int Max(int a, int b)
{
	return (a < b)
		? b
		: a;
}

void Operate(double* firstDevice, double* secondDevice, double* resultDevice, int vectorLength)
{
	for (int i = 0; i < vectorLength - 1; i++)
	{
		resultDevice[i] = Min(firstDevice[i], secondDevice[i]);
	}
}

double Min(double a, double b)
{
	return (a > b)
		? b
		: a;
}