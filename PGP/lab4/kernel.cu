#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include <thrust/extrema.h>
#include <thrust/device_vector.h>

using namespace thrust;

const double EPSILON = 10E-8;

struct Comparator
{
	__host__ __device__ bool operator()(double a, double b)
	{
		return fabs(a) < fabs(b) && fabs(b) >= 10E-8;
	}
};

__global__ void SwapRows(double* deviceMatrix,
	int currentRow,
	int otherRow,
	int currentColumn,
	int rowCount,
	int columnCount)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetx = blockDim.x * gridDim.x;

	for (auto i = idx; i < columnCount; i += offsetx)
	{
		auto temp = deviceMatrix[i * rowCount + currentRow];
		deviceMatrix[i * rowCount + currentRow] = deviceMatrix[i * rowCount + otherRow];
		deviceMatrix[i * rowCount + otherRow] = temp;
	}
}

__global__ void CalculateCurrentRow(double* deviceMatrix,
	int currentRow,
	int currentColumn,
	int rowCount,
	int columnCount)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x + currentColumn + 1;
	int offsetx = blockDim.x * gridDim.x;

	for (auto i = idx; i < columnCount; i += offsetx)
	{
		deviceMatrix[i * rowCount + currentRow] /= deviceMatrix[currentColumn * rowCount + currentRow];
	}
}

__global__ void CalculateRows(double* deviceMatrix, int rowCount, int columnCount, int currentRow, int currentColumn)
{
	int idx = threadIdx.x;
	int offsetx = blockDim.x;
	int idy = blockIdx.x;
	int offsety = gridDim.x;

	for (int j = idx + currentRow + 1; j < rowCount; j += offsetx) {
		for (int k = idy + currentColumn + 1; k < columnCount; k += offsety) {
			deviceMatrix[k * rowCount + j] -= deviceMatrix[currentColumn * rowCount + j] * deviceMatrix[k * rowCount + currentRow];
		}
	}
}

__global__ void SetCurrentZero(double* deviceMatrix,
	int currentRow,
	int currentColumn,
	int rowCount,
	int columnCount)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offsetx = blockDim.x * gridDim.x;

	for (auto i = idx; i < columnCount; i += offsetx)
	{
		deviceMatrix[i * rowCount + currentRow] = 0;
		deviceMatrix[currentColumn * rowCount + i] = 0;
	}
}

Comparator comparator;

__host__ int GetMaxIndexInColumn(double* deviceMatrix,
	int rowIndex,
	int columnIndex,
	int rowCount,
	int columnCount)
{

	if (columnIndex * rowCount == 0)
	{
		auto indexPointer = device_pointer_cast(deviceMatrix + columnIndex * rowCount);
		auto maxIndexPointer = max_element(indexPointer + rowIndex, indexPointer + rowCount, comparator);
		auto maxIndex = maxIndexPointer - indexPointer;

		double elem;
		hipMemcpy(&elem, deviceMatrix + columnIndex * rowCount + maxIndex, sizeof(double), hipMemcpyDeviceToHost);

		if (fabs(elem) < EPSILON)
		{
			return -1;
		}

		return maxIndex;
	}
	else
	{
		auto indexPointer = device_pointer_cast(deviceMatrix + columnIndex * rowCount);
		auto maxIndexPointer = thrust::max_element(indexPointer - 1 + rowIndex, indexPointer + rowCount, comparator);
		auto maxIndex = maxIndexPointer - indexPointer;
		if (maxIndex == rowIndex - 1)
		{
			return -1;
		}

		return maxIndex;
	}
}

__host__ int FindRank(double* matrix, int rowCount, int columnCount)
{
	double* deviceMatrix;
	hipMalloc(&deviceMatrix, rowCount * columnCount * sizeof(double));
	hipMemcpy(deviceMatrix, matrix, rowCount * columnCount * sizeof(double), hipMemcpyHostToDevice);

	auto offset = 0;


	for (int i = 0; i < rowCount && i + offset < columnCount; i++)
	{
		try
		{
			auto maxIndex = GetMaxIndexInColumn(deviceMatrix, i, i + offset, rowCount, columnCount);

			if (maxIndex < 0)
			{
				offset++;
				i--;
				continue;
			}

			if (maxIndex != i)
			{
				SwapRows << <1024, 1024 >> > (deviceMatrix, i, maxIndex, i + offset, rowCount, columnCount);
			}

			CalculateCurrentRow << <1024, 1024 >> > (deviceMatrix, i, i + offset, rowCount, columnCount);
			CalculateRows << <1024, 1024 >> > (deviceMatrix, rowCount, columnCount, i, i + offset);
			SetCurrentZero << <1024, 1024 >> > (deviceMatrix, i, i + offset, rowCount, columnCount);
		}
		catch (std::runtime_error& e)
		{
			//std::cerr << rowCount << ' ' << columnCount << '\n';
			//std::cerr << "offset: " << offset << '\n';
			//std::cerr << e.what() << '\n';
		}
	}

	hipFree(deviceMatrix);

	auto rank = columnCount - offset > rowCount
		? rowCount
		: columnCount - offset;

	return rank;
}

int main()
{
	std::ios_base::sync_with_stdio(false);
	std::cin.tie(nullptr);

	int rowCount, columnCount;
	std::cin >> rowCount >> columnCount;


	auto isTransposed = rowCount < columnCount;

	if (isTransposed)
	{
		auto temp = rowCount;
		rowCount = columnCount;
		columnCount = temp;
	}

	auto matrix = new double[rowCount * columnCount];

	for (int i = 0; i < rowCount; i++)
	{
		for (int j = 0; j < columnCount; j++)
		{
			if (isTransposed)
			{
				std::cin >> matrix[i * columnCount + j];
				//matrix[i * columnCount + j] = rand() % 200 - 100;
			}
			else
			{
				std::cin >> matrix[j * rowCount + i];
				//matrix[j * rowCount + i] = rand() % 200 - 100;
			}
		}
	}

	auto rank = FindRank(matrix, rowCount, columnCount);
	std::cout << rank << std::endl;

	delete[] matrix;
}
